#include "hip/hip_runtime.h"
/*BSD License

Copyright © belongs to the uploader, all rights reserved.

Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

Redistributions of source code must retain the above copyright notice, with the name of the uploader, and this list of conditions;

Redistributions in binary form must reproduce the above copyright notice, with the name of the uploader, and this list of conditions in the documentation and/or other materials provided with the distribution;
Neither the name of the uploader nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <sys/time.h>
// includes, project
#include <cutil.h>
//#include "mycutil.h"

// includes, kernels
#include <parser_kernel.cu>
#include <hash_funcs.h>
#include <defs.h>
#include <list>
int *packet_start_token; // we need as many packets as the number of threads
int *send_tokens_count, *docs_count_arg;
int *packet_doc_map, *doc_size;
float *magnitude_array, *magnitude_res_array;
float *similarity_matrix, *similarity_res_matrix;

extern char cwd[1024];

extern std::list<char*> docs;
/********************************************************/
//MyHashMapElement **freq_packets_array_remote;
//MyHashMapElement **doc_token_hashtable_remote;  // each doc has its own token hash table
MyHashMapElement *occ_hash_table_remote;
int *token_doc_map_remote;
CalcFreqController  *token_division_controller_remote;
CalcFreqController  *token_division_controller_host;
float *doc_similarity_matrix_remote;
float *doc_similarity_matrix_host;
int *doc_rank_matrix_remote;
int *doc_rank_matrix_host;

struct timeval memcpy_start;
struct timeval memcpy_end;
struct timeval parser_start;
struct timeval parser_end;
struct timeval corpus_start;
struct timeval corpus_end;
struct timeval normalize_start;
struct timeval normalize_end;

void create_remote_hash_tables(MyHashMapElement **hash_doc_token_tables, MyHashMapElement **hash_doc_token_sub_tables, int docs_count, int *sub_table_size, int *table_size, int *occ_table_size);

void free_remote_hash_tables(MyHashMapElement **hash_doc_token_tables, MyHashMapElement **hash_doc_token_sub_tables, int docs_count);
void free_local_buffers();

struct timeval profile_start;
struct timeval profile_end;
struct timeval prep_start;
struct timeval prep_end;

long calcDiffTime(struct timeval* strtTime, struct timeval* endTime)
{
    return(
        endTime->tv_sec*1000000 + endTime->tv_usec
        - strtTime->tv_sec*1000000 - strtTime->tv_usec
        );
  
}


void load_parser_kernel(char *token_array, int tokens_count, int *doc_token_count, int docs_count)
{
    CUT_DEVICE_INIT(1, NULL);
    
    printf("Running kernel, cnt = %d \n", tokens_count);

    // allocate host memory for the string tokens
    char *host_local, *host_res;
    unsigned int *token_length_host;
    int *token_doc_map_local;
    host_local = token_array; //(char *)malloc(32*tokens_count*sizeof(char *));
    token_length_host = (unsigned int *)malloc(tokens_count*sizeof(unsigned int));
    token_doc_map_local = (int *)malloc(tokens_count * sizeof(int));
    token_division_controller_host = (CalcFreqController *)malloc(docs_count * sizeof(CalcFreqController));
    doc_similarity_matrix_host = (float *)malloc(docs_count * docs_count * sizeof(float));
    doc_rank_matrix_host = (int *)malloc(docs_count * docs_count * sizeof(int));
    int num_tokens = 0;
    for (int i = 0; i != docs_count; i++)
      {
        token_division_controller_host[i].doc_index = i;
        token_division_controller_host[i].doc_token_start = num_tokens;
        token_division_controller_host[i].doc_token_count = doc_token_count[i];
        num_tokens += doc_token_count[i];
        printf("token_start = %d, token_count = %d\n", token_division_controller_host[i].doc_token_start,
               token_division_controller_host[i].doc_token_count);
      }

    int remain_doc_tokens = doc_token_count[0];
    int cur_doc = 0;
    gettimeofday(&prep_start, NULL); 

    for(int i=0; i<tokens_count; i++)
    {
        int j;
		for(j=0; j<TOKEN_MAX_SIZE_PLUS_END; j++)
		{
            token_array[i * TOKEN_MAX_SIZE_PLUS_END + j] = token_array[i * TOKEN_MAX_SIZE_PLUS_END + j];
			if(token_array[i * TOKEN_MAX_SIZE_PLUS_END + j]=='\0')
				break;
		}
        
        token_length_host[i] = j;
        token_doc_map_local[i] = cur_doc;
        remain_doc_tokens--;
        if (remain_doc_tokens == 0){
          cur_doc++;
          if (i != tokens_count - 1)
            remain_doc_tokens = doc_token_count[cur_doc];
        }
	}
    assert(remain_doc_tokens == 0);
    assert(cur_doc == docs_count);

    gettimeofday(&prep_end, NULL); 
    long prep_time = calcDiffTime(&prep_start, &prep_end);
    printf("prep token time = %ld\n", prep_time);

	
	// allocate device memory
    char *dev_mem;
    CUDA_SAFE_CALL(hipMalloc((void**) &dev_mem, 32*tokens_count*sizeof(char *)));
    unsigned int *token_length_array_mem;
    CUDA_SAFE_CALL(hipMalloc((void**) &token_length_array_mem, tokens_count*sizeof(unsigned int)));
    CUDA_SAFE_CALL(hipMalloc((void**) &token_doc_map_remote, tokens_count*sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**) &token_division_controller_remote, docs_count*sizeof(CalcFreqController)));

    // copy host memory to device
     gettimeofday(&memcpy_start, NULL); 
    CUDA_SAFE_CALL(hipMemcpy(dev_mem, host_local, 32*sizeof(char)*tokens_count, hipMemcpyHostToDevice) );	
    CUDA_SAFE_CALL(hipMemcpy(token_length_array_mem, token_length_host, sizeof(unsigned int)*tokens_count, hipMemcpyHostToDevice) );	
    CUDA_SAFE_CALL(hipMemcpy(token_doc_map_remote, token_doc_map_local, sizeof(int)*tokens_count, hipMemcpyHostToDevice) );	
    CUDA_SAFE_CALL(hipMemcpy(token_division_controller_remote, token_division_controller_host, sizeof(CalcFreqController)*docs_count, hipMemcpyHostToDevice) );	
    gettimeofday(&memcpy_end, NULL); 
        long memcpytime = calcDiffTime(&memcpy_start, &memcpy_end);
    printf("memcpy = %ld\n", memcpytime);

    MyHashMapElement *hash_doc_token_sub_tables_host[MAX_GRID_SIZE];
    MyHashMapElement *hash_doc_token_tables_host[docs_count];
    int sub_table_size, table_size, occ_table_size;
    create_remote_hash_tables(hash_doc_token_tables_host, hash_doc_token_sub_tables_host, docs_count, &sub_table_size, &table_size, &occ_table_size);
    MyHashMapElement **hash_doc_token_sub_tables_remote;
    MyHashMapElement **hash_doc_token_tables_remote;
    CUDA_SAFE_CALL(hipMalloc((void**) &hash_doc_token_sub_tables_remote, MAX_GRID_SIZE * sizeof(MyHashMapElement *)));
    CUDA_SAFE_CALL(hipMalloc((void**) &hash_doc_token_tables_remote, docs_count * sizeof(MyHashMapElement *)));
    CUDA_SAFE_CALL(hipMemcpy(hash_doc_token_sub_tables_remote, hash_doc_token_sub_tables_host, MAX_GRID_SIZE * sizeof(MyHashMapElement *), hipMemcpyHostToDevice));	
    CUDA_SAFE_CALL(hipMemcpy(hash_doc_token_tables_remote, hash_doc_token_tables_host, docs_count * sizeof(MyHashMapElement *), hipMemcpyHostToDevice)); 


//    CUDA_SAFE_CALL(hipMemcpy((send_tokens_count), &tokens_count, sizeof(int), hipMemcpyHostToDevice) );
    
    // create and start timer
    unsigned int timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    CUT_SAFE_CALL(cutStartTimer(timer));

    // setup execution parameters
    dim3 dimBlock(MAX_THREADS);
    dim3 dimGrid(docs_count); // TODO floor(tokens_count/(threads.x * 2)));


    gettimeofday(&profile_start, NULL); 

    gettimeofday(&parser_start, NULL); 
    StripAffixes<<< dimGrid, dimBlock >>>(dev_mem, token_length_array_mem, token_division_controller_remote); //send_tokens_count);
    dbg
      { 
        CUT_SAFE_CALL(cutStopTimer(timer));
        printf("\nStrip affixes time: %f (ms) n\n", cutGetTimerValue(timer));

        host_res = (char *)malloc(TOKEN_MAX_SIZE_PLUS_END * tokens_count * sizeof(char));
        CUDA_SAFE_CALL(hipMemcpy(host_res, dev_mem, TOKEN_MAX_SIZE_PLUS_END*sizeof(char)*tokens_count, hipMemcpyDeviceToHost) );
        CUDA_SAFE_CALL(hipMemcpy(token_length_host, token_length_array_mem, sizeof(unsigned int)*tokens_count, hipMemcpyDeviceToHost) );
        for(int i=0; i<tokens_count; i++)
    		{
    			for(int j=0; j<TOKEN_MAX_SIZE_PLUS_END; j++)
    			{
    				if(host_res[i*TOKEN_MAX_SIZE_PLUS_END+j]=='\0')
    					break;
    				//printf("%c",host_res[i*32+j]);
    			}
    			printf("\n%3d %s %s (%d %d)",i, &token_array[i * TOKEN_MAX_SIZE_PLUS_END], &host_res[i*TOKEN_MAX_SIZE_PLUS_END], token_length_host[i], token_doc_map_local[i]);
    		}
        CUT_SAFE_CALL(cutStartTimer(timer));
      }

    InitOccTable<<<OCC_HASH_TABLE_SIZE/32, 32>>>(occ_hash_table_remote);  // TODO make it multi-grid
    dimBlock.x = HASH_DOC_TOKEN_NUM_THREADS;
    for (int i = 0; i != docs_count;) // TODO we can do only one batch
      { 
        dimGrid.x = min(16, docs_count - i);  // TODO replace the magic number

        MakeDocHash<<< dimGrid, dimBlock >>>(dev_mem, token_length_array_mem, &token_division_controller_remote[i], hash_doc_token_sub_tables_remote, &hash_doc_token_tables_remote[i], sub_table_size, table_size);
        
        i += dimGrid.x;
      }
    gettimeofday(&parser_end, NULL); 
    long parsetime = calcDiffTime(&parser_start, &parser_end);
    printf("parsetime = %ld\n", parsetime);


    gettimeofday(&corpus_start, NULL); 
    assert(HASH_DOC_TOKEN_TABLE_SIZE % 32 == 0);
    AddToOccTable<<<HASH_DOC_TOKEN_TABLE_SIZE/32, 32>>>(hash_doc_token_tables_remote, occ_hash_table_remote, docs_count);
    gettimeofday(&corpus_end, NULL); 
    long corpustime = calcDiffTime(&corpus_start, &corpus_end);
    printf("corpustime = %ld\n", corpustime);

    dimBlock.x = HASH_DOC_TOKEN_TABLE_SIZE;
    dimGrid.x = docs_count;
    
    gettimeofday(&normalize_start, NULL); 
    float *bucket_sqrt_sum_remote;
    CUDA_SAFE_CALL(hipMalloc((void**) &bucket_sqrt_sum_remote, docs_count * HASH_DOC_TOKEN_TABLE_SIZE * sizeof(float)));
    CalcTfIdf<<<dimGrid, dimBlock>>>(token_division_controller_remote, hash_doc_token_tables_remote, occ_hash_table_remote, docs_count, bucket_sqrt_sum_remote);

    dimBlock.x = 1;

    CalcTfIdf2<<<dimGrid, dimBlock>>>(token_division_controller_remote, hash_doc_token_tables_remote, occ_hash_table_remote, docs_count, bucket_sqrt_sum_remote);

    dimBlock.x = HASH_DOC_TOKEN_TABLE_SIZE;

    CalcTfIdf3<<<dimGrid, dimBlock>>>(token_division_controller_remote, hash_doc_token_tables_remote, occ_hash_table_remote, docs_count, bucket_sqrt_sum_remote);
    gettimeofday(&normalize_end, NULL); 
    long tfidftime = calcDiffTime(&normalize_start, &normalize_end);
    printf("tfidf = %ld\n", tfidftime);

    dimGrid.x = docs_count;    dimGrid.y = docs_count;
    dimBlock.x = HASH_DOC_TOKEN_TABLE_SIZE; 
    // each block does a pair similarity
    CalcSimilarities<<< dimGrid, dimBlock >>>(hash_doc_token_tables_remote, occ_hash_table_remote, doc_similarity_matrix_remote, docs_count);
    dimGrid.x = docs_count ;   dimGrid.y = 1;
    dimBlock.x = docs_count;
    SortSimilarities<<< dimGrid, dimBlock >>>(doc_similarity_matrix_remote, doc_rank_matrix_remote, docs_count);

    gettimeofday(&profile_end, NULL);
    long profile_time = calcDiffTime(&profile_start, &profile_end);
    printf("total kernel time = %ld\n", profile_time);

        //        CalcIDF
    dbg{
       CUT_SAFE_CALL(cutStopTimer(timer));
       printf("\nHash Doc table time: %f (ms) n\n", cutGetTimerValue(timer));
       for (int i = 0 ; i != 16; i++)
         printf("subtable %d address 0x%x\n", i, hash_doc_token_sub_tables_host[i]);

       //       CUDA_SAFE_CALL(hipMemcpy(token_length_host, token_length_array_mem, sizeof(unsigned int)*tokens_count, hipMemcpyDeviceToHost) );
       //       for (int i = 0; i != dimBlock.x; i++)
       //         printf("thread %d's sub table address = 0x%x.\n", i ,token_length_host[i]);

       MyHashMapElement *tables_host[docs_count];
       int doc = 39; if (doc < docs_count)//for (int doc = 39; doc != docs_count; doc+)
         {
           tables_host[doc] = (MyHashMapElement *)malloc(table_size * sizeof (MyHashMapElement));
           CUDA_SAFE_CALL(hipMemcpy(tables_host[doc], hash_doc_token_tables_host[doc], table_size*sizeof(MyHashMapElement), hipMemcpyDeviceToHost) );
           printf ("The %d'th docuemnt hash table:\n", doc);
           MyHashMapElement *table = tables_host[doc];
           for (int j = 0; j != HASH_DOC_TOKEN_TABLE_SIZE; j++)
             {
               printf("The %d'th document hash table, the %d'th bucket\n", doc, j);
               for (int ele = 0; ele != HASH_DOC_TOKEN_BUCKET_SIZE; ele++)
                 {
                   printf("count in bucket(%d), key(0x%x),freq(%d), tokenLen(%d),subkey(%d) tfidf(%f) \n", table[ele].countInBuc,
                          table[ele].key, table[ele].freq, table[ele].tokenLength, table[ele].subkey, table[ele].tfidf);
                 }
               table += HASH_DOC_TOKEN_BUCKET_SIZE;
             }

           free(tables_host[doc]);
         }
       MyHashMapElement *occ_table_host;
       occ_table_host = (MyHashMapElement *)malloc(occ_table_size * sizeof (MyHashMapElement));
       CUDA_SAFE_CALL(hipMemcpy(occ_table_host, occ_hash_table_remote, occ_table_size*sizeof(MyHashMapElement), hipMemcpyDeviceToHost) );
       printf("occurence table\n");
       for (int occ = 0; occ != OCC_HASH_TABLE_SIZE; occ++)
         {
           MyHashMapElement *bucket = &occ_table_host[occ * OCC_HASH_TABLE_BUCKET_SIZE];
           printf("occurrence table: the %d'th bucket:\n", occ);
           for (int ele = 0; ele != OCC_HASH_TABLE_BUCKET_SIZE; ele++)
             {
               printf("count in bucket(%d), key(0x%x),freq(%d), tokenLen(%d),subkey(%d) \n", bucket[ele].countInBuc,
                      bucket[ele].key, bucket[ele].freq, bucket[ele].tokenLength, bucket[ele].subkey);
             }
         }

    }
    CUDA_SAFE_CALL(hipMemcpy(doc_similarity_matrix_host, doc_similarity_matrix_remote, docs_count * docs_count * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(doc_rank_matrix_host, doc_rank_matrix_remote, docs_count * docs_count * sizeof(int), hipMemcpyDeviceToHost));    
    dbg {
      printf("similarity matrix: \n");
      for (int doc1 = 0; doc1 != docs_count; doc1++)
        {
          for (int doc2 = 0; doc2 != docs_count; doc2++)
            printf("%5f(%d) ", doc_similarity_matrix_host[doc1*docs_count + doc2], doc_rank_matrix_host[doc1*docs_count + doc2]);
          printf("\n");
        }
    }

    float *sim = doc_similarity_matrix_host;
    int *rank = doc_rank_matrix_host;
    std::list<char*>::const_iterator doc1i = docs.begin();
    for (int doc1 = 0; doc1 != docs_count; doc1++, doc1i++)
      {
        printf("\n%s : \n", &(*doc1i)[strlen(cwd)]);

        for (int r = 0; r != 10; r++)
          {
            int find = 0;
            std::list<char*>::const_iterator doc2i = docs.begin();
            for (int doc2 = 0; doc2 != docs_count; doc2++, doc2i++)
              {
                if (rank[doc1 * docs_count + doc2] == r)
                  {
                    printf("%5f, %s\n", sim[doc1 * docs_count + doc2], &(*doc2i)[strlen(cwd)]);
                    find = 1;
                  }
              }
            if (!find) break;
          }
      }
    

    CUT_CHECK_ERROR("Kernel execution failed");

	CUT_SAFE_CALL(cutDeleteTimer(timer));

    CUDA_SAFE_CALL(hipFree(dev_mem));
    free_remote_hash_tables(hash_doc_token_tables_host, hash_doc_token_sub_tables_host, docs_count);
    free_local_buffers();    
    CUT_EXIT(0, 0);

}

void create_remote_hash_tables(MyHashMapElement **hash_doc_token_tables, MyHashMapElement **hash_doc_token_sub_tables, int docs_count, int *sub_table_size, int *table_size, int *occ_table_size)
{
  *sub_table_size = HASH_DOC_TOKEN_SUB_TABLE_SIZE*HASH_DOC_TOKEN_NUM_THREADS* HASH_DOC_TOKEN_BUCKET_SUB_SIZE;
  *table_size = HASH_DOC_TOKEN_TABLE_SIZE * HASH_DOC_TOKEN_BUCKET_SIZE;
  *occ_table_size = OCC_HASH_TABLE_SIZE * OCC_HASH_TABLE_BUCKET_SIZE;
  for (int i = 0; i != MAX_GRID_SIZE; i++)
    {
      CUDA_SAFE_CALL(hipMalloc((void **)&hash_doc_token_sub_tables[i], (*sub_table_size)*sizeof(MyHashMapElement)));
    }
  for (int i = 0; i != docs_count; i++)
    {
      CUDA_SAFE_CALL(hipMalloc((void **)&hash_doc_token_tables[i], (*table_size)*sizeof(MyHashMapElement)));
    }
  CUDA_SAFE_CALL(hipMalloc((void **)&occ_hash_table_remote, (*occ_table_size) * sizeof(MyHashMapElement)));

  CUDA_SAFE_CALL(hipMalloc((void **)&doc_similarity_matrix_remote, docs_count * docs_count * sizeof(float)));
  CUDA_SAFE_CALL(hipMalloc((void **)&doc_rank_matrix_remote, docs_count * docs_count * sizeof(float)));

  printf("Allocating remote memory size = %d K bytes for hash_token_sub_tables\n", (*sub_table_size)*sizeof(MyHashMapElement) * docs_count/1024);
  printf("Allocating remote memory size = %d K bytes for hash_token_tables.\n", (*table_size)*sizeof(MyHashMapElement) * docs_count / 1024);
  printf("Allocating remote memory size = %d K bytes for global occurence table.\n", (*occ_table_size) * sizeof(MyHashMapElement)/1024);
}

void free_local_buffers()
{
  free(doc_similarity_matrix_host);
}

void free_remote_hash_tables(MyHashMapElement **hash_doc_token_tables, MyHashMapElement **hash_doc_token_sub_tables, int docs_count)
{
  for (int i = 0; i != MAX_GRID_SIZE; i++)
      CUDA_SAFE_CALL(hipFree(hash_doc_token_sub_tables[i]));

  for (int i = 0; i != docs_count; i++)
    CUDA_SAFE_CALL(hipFree(hash_doc_token_tables[i]));

  CUDA_SAFE_CALL(hipFree(occ_hash_table_remote));
  CUDA_SAFE_CALL(hipFree(doc_similarity_matrix_remote));
  CUDA_SAFE_CALL(hipFree(doc_rank_matrix_remote));
}


