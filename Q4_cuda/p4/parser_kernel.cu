#include "hip/hip_runtime.h"
/*BSD License

Copyright © belongs to the uploader, all rights reserved.

Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

Redistributions of source code must retain the above copyright notice, with the name of the uploader, and this list of conditions;

Redistributions in binary form must reproduce the above copyright notice, with the name of the uploader, and this list of conditions in the documentation and/or other materials provided with the distribution;
Neither the name of the uploader nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
*/

#ifndef _PARSER_KERNEL_H_
#define _PARSER_KERNEL_H_

#include <stdio.h>
#include <string.h>
#include "string_funcs.cu"
#include "hash_funcs.cu"
#include "defs.h"


__device__ bool stripPrefixes ( char *str);

__device__ char prefixes[][16]= { "kilo", "micro", "milli", "intra", "ultra", "mega", "nano", "pico", "pseudo"};

__device__ char suffixes2[][2][16] = { { "ational", "ate" },
            { "tional",  "tion" },
            { "enci",    "ence" },
            { "anci",    "ance" },
            { "izer",    "ize" },
            { "iser",    "ize" },
            { "abli",    "able" },
            { "alli",    "al" },
            { "entli",   "ent" },
            { "eli",     "e" },
            { "ousli",   "ous" },
            { "ization", "ize" },
            { "isation", "ize" },
            { "ation",   "ate" },
            { "ator",    "ate" },
            { "alism",   "al" },
            { "iveness", "ive" },
            { "fulness", "ful" },
            { "ousness", "ous" },
            { "aliti",   "al" },
            { "iviti",   "ive" },
            { "biliti",  "ble" }};

__device__ char suffixes3[][2][16] = { { "icate", "ic" },
            { "ative", "" },
            { "alize", "al" },
            { "alise", "al" },
            { "iciti", "ic" },
            { "ical",  "ic" },
            { "ful",   "" },
            { "ness",  "" }};
            
__device__ char suffixes4[][16] = { "al",
            "ance",
            "ence",
            "er",
            "ic",
            "able", "ible", "ant", "ement", "ment", "ent", "sion", "tion",
            "ou", "ism", "ate", "iti", "ous", "ive", "ize", "ise"};


__device__ bool step1(char *str ) {

    char stem[32];
    bool changed = false;
    if ( str[strLen(str)-1] == 's' ) {
        if ( (hasSuffix( str, "sses", stem ))
                || (hasSuffix( str, "ies", stem)) ){
            str[strLen(str)-2] = '\0';
            changed = true;
        }
        else {
            if ( ( strLen(str) == 1 )
                    && ( str[strLen(str)-1] == 's' ) ) {
                str[0] = '\0';
                return true;
            }
            if ( str[strLen(str)-2 ] != 's' ) {
                str[strLen(str)-1] = '\0';
                changed = true;
            }
        }
    }

    if ( hasSuffix( str,"eed",stem ) ) {
        if ( measure( stem ) > 0 ) {
            str[strLen(str)-1] = '\0';
            changed = true;
        }
    }
    else {
        if (  (hasSuffix( str,"ed",stem ))
                || (hasSuffix( str,"ing",stem )) ) {
            if (containsVowel( stem ))  {

                if(stem[0]=='\0')
                  {
					str[0]='\0';
                    changed = true;
                  }
				else
                  {
					str[strLen(stem)] = '\0';
                    changed = true;
                  }
                if ( strLen(str) == 1 )
                    return changed;

                if ( ( hasSuffix( str,"at",stem) )
                        || ( hasSuffix( str,"bl",stem ) )
                        || ( hasSuffix( str,"iz",stem) ) ) {
                    int len = strLen(str);
                    str[len-1] = 'e';
                    str[len] = '\0';
                    changed = true;

                }
                else {
                    int length = strLen(str);
                    if ( (str[length-1] == str[length-2])
                            && (str[length-1] != 'l')
                            && (str[length-1] != 's')
                            && (str[length-1] != 'z') ) {
                        str[length-1]='\0';
                        changed = true;
                    }
                    else
                    if ( measure( str ) == 1 ) {
                        if ( cvc(str) )
                        {
                           str[length-1]='e';
                           str[length]='\0';
                           changed = true;
                        }   
                    }
                }
            }
        }
    }

    if ( hasSuffix(str,"y",stem) )
        if ( containsVowel( stem ) ) {
            int len = strLen(str);
            str[len-1]='i';
            str[len]='\0';
            changed = true;
        }
    return changed;
}

__device__ bool step2( char *str ) {
    
    char stem[32];
    int last = sizeof(suffixes2)/(sizeof(char)*2*16); //strange way of calculating length of array
    bool changed = false;

    for ( int index = 0 ; index < last; index++ ) {
        if ( hasSuffix ( str, suffixes2[index][0], stem ) ) {
            if ( measure ( stem ) > 0 ) {
                int stemlen, suffixlen, j;
                stemlen = strLen(stem);
                suffixlen = strLen(suffixes2[index][1]);
                changed = true;
                for(j=0; j<suffixlen; j++)
					str[stemlen+j] = suffixes2[index][1][j];
				str[stemlen+j] = '\0';
            }
        }
    }
    return changed;
}

__device__ bool step3( char *str ) {

    char stem[32];
    int last = sizeof(suffixes3)/(sizeof(char)*2*16); //strange way of calculating length of array/    
    bool changed= false;
    for ( int index = 0 ; index<last; index++ ) {
        if ( hasSuffix ( str, suffixes3[index][0], stem ))
            if ( measure ( stem ) > 0 ) {
                int stemlen, suffixlen, j;
                stemlen = strLen(stem);
                suffixlen = strLen(suffixes3[index][1]);
                changed = true;
                for( j=0; j<suffixlen; j++)
					str[stemlen+j] = suffixes3[index][1][j];
				str[stemlen+j] = '\0';
            }
    }
    return changed ;  
}

__device__ bool step4( char *str ) {

    char stem[32];
    int last = sizeof(suffixes4)/(sizeof(char)*16); //strange way of calculating length of array
    bool changed = false;
    for ( int index = 0 ; index<last; index++ ) {
        if ( hasSuffix ( str, suffixes4[index], stem ) ) {
            changed = true;
            if ( measure ( stem ) > 1 ) {
                str[strLen(stem)] = '\0';
            }
        }
    }
    return changed;
}

__device__ bool step5( char *str ) {

  bool changed = false;
    if ( str[strLen(str)-1] == 'e' ) {
        if ( measure(str) > 1 ) {
            str[strLen(str)-1] = '\0';
            changed = true;
        }
        else
        if ( measure(str) == 1 ) {
            char stem[32];
            int i;
            for ( i=0; i<strLen(str)-1; i++ )
                stem[i] = str[i];
            stem[i] = '\0';

            if ( !cvc(stem) ){
                str[strLen(str)-1] = '\0';
                changed = true;
            }
        }
    }

    if ( strLen(str) == 1 )
        return true;
    if ( (str[strLen(str)-1] == 'l')
            && (str[strLen(str)-2] == 'l') && (measure(str) > 1) )
        if ( measure(str) > 1 ) {
            str[strLen(str)-1] = '\0';
            changed = true;
        }
        
    return changed;
}



__device__ bool stripSuffixes(char *str ) {

  bool changed = false;
    changed = step1( str );
    if ( strLen(str) >= 1 )
        changed |= step2( str );
    if ( strLen(str) >= 1 )
        changed |= step3( str );
    if ( strLen(str) >= 1 )
        changed |= step4( str );
    if ( strLen(str) >= 1 )
        changed |= step5( str );
    return changed;
}

__device__ bool stripPrefixes ( char *str) {

    int  newLen, j;
    bool found = false;

    int last = sizeof(prefixes)/(sizeof(char)*16); //strange way of calculating length of array
    for ( int i=0 ; i<last; i++ ) 
    {
        //Find if str starts with prefix prefixes[i]
        found = prefixFind(str, prefixes[i]);
        if (found)
        {
            newLen = strLen(str) - strLen(prefixes[i]);
            for (j=0 ; j < newLen; j++ )
                str[j] = str[j+strLen(prefixes[i])];
            str[j] = '\0';
        }
    }
    return found;
}


__global__ void
StripAffixes(char *dev_res, unsigned int *token_length, CalcFreqController *controller)
{
     // add __shared__ for operations in str in loop below
     // adjust the token and token_length array pointer according to controller 
    char *base = &dev_res[controller[blockIdx.x].doc_token_start * TOKEN_MAX_SIZE_PLUS_END];
    unsigned int *token_length_base = &token_length[controller[blockIdx.x].doc_token_start];

    int tokens_count = controller[blockIdx.x].doc_token_count;
  	int step_count = tokens_count/blockDim.x;
    int remain = tokens_count - step_count * blockDim.x;
    int index = threadIdx.x *  TOKEN_MAX_SIZE_PLUS_END;
    if (threadIdx.x < remain )
      step_count += 1;

    __shared__ int *str[MAX_THREADS];
    int step_size = blockDim.x * TOKEN_MAX_SIZE_PLUS_END;

    for(int i=0; i< step_count; i++, index+=step_size) {
      str[threadIdx.x] = (int *)&base[index];
      bool changed = ToLowerCase( (char *)str[threadIdx.x]);
      changed |= Clean( (char *)str[threadIdx.x]);
      changed |= stripPrefixes((char *)str[threadIdx.x]);
      changed |= stripSuffixes((char *)str[threadIdx.x]);
      if (changed){
        token_length_base[index/TOKEN_MAX_SIZE_PLUS_END] = strLen((char *)str[threadIdx.x]);
      }	
    }
    return;
}

__global__ void 
InitOccTable(MyHashMapElement *occ_hash_table)
{
  MyHashMapElement *bucket = &occ_hash_table[((blockIdx.x * blockDim.x ) + threadIdx.x) * OCC_HASH_TABLE_BUCKET_SIZE];
  bucket->countInBuc = 0;
  dbg{
    bucket->key = 0xDEADBEEF;
    bucket->freq = 0;
    bucket->tokenLength = 0;
    bucket->subkey = 0;
    for (int j = 1; j < OCC_HASH_TABLE_BUCKET_SIZE; j++)
      {
        bucket[j].countInBuc = 0; 
        bucket[j].key = 0xDEADBEEF;
        bucket[j].freq = 0;
        bucket[j].tokenLength = 0;
        bucket[j].subkey = 0;
      }
  }
}

__global__ void
MakeDocHash(char *dev_mem, unsigned int *token_length, CalcFreqController *controller, 
         MyHashMapElement **hash_doc_token_sub_tables, MyHashMapElement **hash_doc_token_tables, int sub_table_size, int table_size)
{
    char *token_base = &dev_mem[controller[blockIdx.x].doc_token_start * TOKEN_MAX_SIZE_PLUS_END];
    unsigned int *token_length_base = &token_length[controller[blockIdx.x].doc_token_start];
    MyHashMapElement *hash_doc_token_sub_table = hash_doc_token_sub_tables[blockIdx.x];
    MyHashMapElement *hash_doc_token_table = hash_doc_token_tables[blockIdx.x];
    hash_doc_token_sub_table += sub_table_size * threadIdx.x / HASH_DOC_TOKEN_NUM_THREADS;
 
    {// clear the doc hash sub table in each thread
      initHashTable(hash_doc_token_sub_table, HASH_DOC_TOKEN_SUB_TABLE_SIZE, HASH_DOC_TOKEN_BUCKET_SUB_SIZE);
   
      // clear the doc hash table
      int bucketsPerThread = HASH_DOC_TOKEN_TABLE_SIZE / blockDim.x;
      if (threadIdx.x < HASH_DOC_TOKEN_TABLE_SIZE % blockDim.x)
        bucketsPerThread += 1;
      
      MyHashMapElement *bucket = &hash_doc_token_table[threadIdx.x * HASH_DOC_TOKEN_BUCKET_SIZE ];
      for (int i = 0; i != bucketsPerThread; i++)
        {
          bucket->countInBuc = 0;
          dbg{
            bucket->key = 0xDEADBEEF;
            bucket->subkey = 0;
            bucket->freq = 0;
            bucket->tokenLength = 0;
            for (int j = 1; j != HASH_DOC_TOKEN_BUCKET_SIZE; j++)
              {
                bucket[j].countInBuc = 0;
                bucket[j].key = 0xDEADBEEF;
                bucket[j].subkey = 0;
                bucket[j].freq = j;
                bucket[j].tokenLength = 0;
              }
          }
          bucket += blockDim.x * HASH_DOC_TOKEN_BUCKET_SIZE;
        }
    }

    int tokens_count = controller[blockIdx.x].doc_token_count;
  	int step_count = tokens_count/blockDim.x;
    int remain = tokens_count - step_count * blockDim.x;
    int index = threadIdx.x *  TOKEN_MAX_SIZE_PLUS_END;
    if (threadIdx.x < remain )
      step_count += 1;

    //    int *str;
    int step_size = blockDim.x * TOKEN_MAX_SIZE_PLUS_END;

	for(int i=0; i< step_count; i++, index+=step_size)
	{
      unsigned long key  = computeHash(&token_base[index]);
      insertElement(hash_doc_token_sub_table, key, HASH_DOC_TOKEN_SUB_TABLE_SIZE_LOG2, HASH_DOC_TOKEN_BUCKET_SUB_SIZE, token_length_base[index/TOKEN_MAX_SIZE_PLUS_END], 1);
    }

    __syncthreads();  // sub table construction is done

    // merge sub tables into one doc hash table
    hash_doc_token_sub_table = hash_doc_token_sub_tables[blockIdx.x];
    hash_doc_token_sub_table += threadIdx.x * HASH_DOC_TOKEN_BUCKET_SUB_SIZE;
    for (int i = 0; i != HASH_DOC_TOKEN_NUM_THREADS; i++)
      {
        MyHashMapElement *bucket = hash_doc_token_sub_table;
        int numInBucket = bucket->countInBuc;
        while(numInBucket--)
          {
            unsigned long key = bucket->key;
            insertElement(hash_doc_token_table, key, HASH_DOC_TOKEN_TABLE_SIZE_LOG2, HASH_DOC_TOKEN_BUCKET_SIZE, bucket->tokenLength, bucket->freq);
            bucket++;
          }
        hash_doc_token_sub_table += HASH_DOC_TOKEN_SUB_TABLE_SIZE * HASH_DOC_TOKEN_BUCKET_SUB_SIZE;
      }
  
  return;
}

__global__ void
AddToOccTable(MyHashMapElement **hash_doc_token_tables, MyHashMapElement *occ_hash_table, int numDocs)
{
  for (int i = 0; i != numDocs; i++)
    {
      MyHashMapElement *hash_doc_token_table = hash_doc_token_tables[i];
      MyHashMapElement *bucket = &hash_doc_token_table[(blockIdx.x * blockDim.x + threadIdx.x) * HASH_DOC_TOKEN_BUCKET_SIZE];
      int numInBucket = bucket->countInBuc;
      while (numInBucket--)
        {
          unsigned long key = bucket->key;
          insertElement(occ_hash_table, key, OCC_HASH_TABLE_SIZE_LOG2, OCC_HASH_TABLE_BUCKET_SIZE, bucket->tokenLength, 1);
          bucket++;
        }
    }
}

__global__ void 
CalcTfIdf(CalcFreqController *controller,  MyHashMapElement **hash_doc_token_tables, MyHashMapElement *occ_hash_table, int docs_count, float *bucket_sqrt_sum)
{
  // add __shared__ for bucket_sqrt_sum within one block
  int token_doc_count = controller[blockIdx.x].doc_token_count;
  int sumindex = blockIdx.x * HASH_DOC_TOKEN_TABLE_SIZE + threadIdx.x;
  // 1. calculate the un-normalized tfidf
  MyHashMapElement *bucket = hash_doc_token_tables[blockIdx.x];
  bucket += threadIdx.x * HASH_DOC_TOKEN_BUCKET_SIZE;
  int numInBucket = bucket->countInBuc;
  float bucketSqrtSum = 0.0f;
  while (numInBucket--)
    {
      unsigned long key = bucket->key;
      int occ = findElement(occ_hash_table, key, OCC_HASH_TABLE_SIZE_LOG2, OCC_HASH_TABLE_BUCKET_SIZE, bucket->tokenLength);
      if (occ != 0)  // we should be able to find it in the occ table
        {
          float tf = (float)bucket->freq/token_doc_count;
          float idf = log(float(docs_count)/occ);
          bucket->tfidf = tf * idf;
          bucketSqrtSum += bucket->tfidf * bucket->tfidf;
          dbg {
            bucket->subkey = occ;
          }
        }
      bucket++;
    }
  bucket_sqrt_sum[sumindex] = bucketSqrtSum;
}

__global__ void 
CalcTfIdf2(CalcFreqController *controller,  MyHashMapElement **hash_doc_token_tables, MyHashMapElement *occ_hash_table, int docs_count, float *bucket_sqrt_sum)
{
  // merge with CalcTfIdf(), use local reduction, add __syncthreads() where needed (and only there)
  int sumindex = blockIdx.x * HASH_DOC_TOKEN_TABLE_SIZE;
  float sum = 0.0f;
    int i;
    for (i = 0; i < HASH_DOC_TOKEN_TABLE_SIZE; i++)
      sum += bucket_sqrt_sum[sumindex + i];
    bucket_sqrt_sum[sumindex] = sqrt(sum);
}

__global__ void 
CalcTfIdf3(CalcFreqController *controller,  MyHashMapElement **hash_doc_token_tables, MyHashMapElement *occ_hash_table, int docs_count, float *bucket_sqrt_sum)
{
  // merge with CalcTfIdf()
  MyHashMapElement *bucket;
  int numInBucket;
  // 3. normalize
  float magnitude = bucket_sqrt_sum[blockIdx.x * HASH_DOC_TOKEN_TABLE_SIZE];
  bucket = hash_doc_token_tables[blockIdx.x];
  bucket += threadIdx.x * HASH_DOC_TOKEN_BUCKET_SIZE;
  numInBucket = bucket->countInBuc;
  while (numInBucket--)
    {
      float tfidf = (float)bucket->tfidf;
      tfidf = tfidf / magnitude;
      bucket->tfidf = tfidf;
      bucket++;
    }
}

__global__ void
CalcSimilarities(MyHashMapElement **hash_doc_token_tables, MyHashMapElement *occ_hash_table_remote, float *similarity_matrix, int docs_count)
{
  //  add __shared__ for similarity over all tokens in one doc, use reduction to write into similarity_matrix in 2nd loop
  MyHashMapElement *hashDoc_token_table1 = hash_doc_token_tables[blockIdx.x]; 
  MyHashMapElement *hashDoc_token_table2 = hash_doc_token_tables[blockIdx.y]; 
  float sim_sum = 0.0f;
  MyHashMapElement *bucket1 = hashDoc_token_table1 + threadIdx.x * HASH_DOC_TOKEN_BUCKET_SIZE;

  int num_ele_1 = bucket1->countInBuc;
  while (num_ele_1--)
    {
      MyHashMapElement *bucket2 = hashDoc_token_table2 + threadIdx.x * HASH_DOC_TOKEN_BUCKET_SIZE;
      int num_ele_2 = bucket2->countInBuc;
      int find = 0;
      while (num_ele_2--)
        {
          if ((bucket2->key == bucket1->key) && (bucket2->tokenLength == bucket1->tokenLength))
            {
              find = 1;
              break;
            }
          bucket2++;
        }
      if (find)
        sim_sum += bucket1->tfidf * bucket2->tfidf;

      bucket1++;
    }

    // 2nd loop
    if (threadIdx.x == 0)
      similarity_matrix[docs_count * blockIdx.x + blockIdx.y] = sim_sum;
    int i;
    for (i = 1; i < HASH_DOC_TOKEN_TABLE_SIZE; i++) {
      __syncthreads();
      if (threadIdx.x == i)
        similarity_matrix[docs_count * blockIdx.x + blockIdx.y] += sim_sum;
    }
}

/* This is only OK for small number of documents
 It returns the position of each entry in sorted pattern.
 On the host, extra work needs to be done to search for intended position. 
 TODO make it faster for large number of documents
*/
__global__ void
SortSimilarities(float *similarity_matrix, int *rank_matrix, int docs_count)
{
  __shared__ float similarity[512];   // TODO max docs count?
  float *sim_base = &similarity_matrix[blockIdx.x * docs_count];
  similarity[threadIdx.x] = sim_base[threadIdx.x];
  __syncthreads();
  
  float my_value = similarity[threadIdx.x];
  int myRank = 0;
  for (int i = 0; i != docs_count; i++)
    {
      if (i == threadIdx.x) 
        continue;
      if (similarity[i] > my_value) 
        myRank++;
    }

  rank_matrix[blockIdx.x * docs_count + threadIdx.x] = myRank;
}

#endif // #ifndef _PARSER_KERNEL_H_

